#include "hip/hip_runtime.h"
/* 
   Programación de GPUs (General Purpose Computation on Graphics Processing 
   Unit)

   Margarita Amor López
   Emilio J. Padrón González

   sumavectores.c

   Suma dos vectores en CPU y GPU
   Parámetros opcionales (en este orden): sumavectores #rep #n #blk
   #rep: número de repetiones
   #n: número de elementos en cada vector
   #blk: hilos por bloque CUDA
*/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>


const int N = 1048576;    // Número predeterm. de elementos en los vectores

const int CUDA_BLK = 1024;  // Tamaño predeterm. de bloque de hilos ƒCUDA


/* 
   Para medir el tiempo transcurrido (elapsed time):

   resnfo: tipo de dato definido para abstraer la métrica de recursos a usar
   timenfo: tipo de dato definido para abstraer la métrica de tiempo a usar

   timestamp: abstrae función usada para tomar las muestras del tiempo transcurrido

   printtime: abstrae función usada para imprimir el tiempo transcurrido

   void myElapsedtime(resnfo start, resnfo end, timenfo *t): función para obtener 
   el tiempo transcurrido entre dos medidas
*/

#include <sys/time.h>
#include <sys/resource.h>

#ifdef _noWALL_
typedef struct rusage resnfo;
typedef struct _timenfo {
  double time;
  double systime;
} timenfo;
#define timestamp(sample) getrusage(RUSAGE_SELF, (sample))
#define printtime(t) printf("%15f s (%f user + %f sys) ",		\
			    t.time + t.systime, t.time, t.systime);
#else
typedef struct timeval resnfo;
typedef double timenfo;
#define timestamp(sample)     gettimeofday((sample), 0)
#define printtime(t) printf("%15f s ", t);
#endif

void myElapsedtime(const resnfo start, const resnfo end, timenfo *const t)
{
#ifdef _noWALL_
  t->time = (end.ru_utime.tv_sec + (end.ru_utime.tv_usec * 1E-6)) 
    - (start.ru_utime.tv_sec + (start.ru_utime.tv_usec * 1E-6));
  t->systime = (end.ru_stime.tv_sec + (end.ru_stime.tv_usec * 1E-6)) 
    - (start.ru_stime.tv_sec + (start.ru_stime.tv_usec * 1E-6));
#else
  *t = (end.tv_sec + (end.tv_usec * 1E-6)) 
    - (start.tv_sec + (start.tv_usec * 1E-6));
#endif /*_noWALL_*/
}


/*
  Función para inicializar los vectores que vamos a utilizar
*/
void populating_arrays(float arrayA[], float arrayB[], 
		       float arrayR[], const unsigned int n)
{
  unsigned int i;

  for(i = 0; i < n; i++) {
    arrayA[i] = i;
    arrayB[i] = n-i;
    arrayR[i] = 0;
  }
}


/*
  Función que devuelve la suma de todos los elementos de un vector, 
  y que usaremos para comprobar el resultado. 
  De paso inicializa el array.
*/
float checkini_array(float array[], const unsigned int n)
{
  unsigned int i;
  float res = 0;

  for(i = 0; i < n; i++) {
    res += array[i];
    array[i] = 0;
  }

  return(res);
}


/*
  Función para sumar dos vectores en la CPU *r* veces
*/
void add_arrays_CPU(const float arrayA[], const float arrayB[], 
		    float arrayR[], const unsigned int n)
{
  unsigned int i;


    for(i = 0; i < n; i++) {
      arrayR[i] += arrayA[i] + arrayB[i];
    }
  
}


// Declación de kernel, definición más abajo
__global__ void add_arrays_kernel_cuda(const float *const mA, 
				       const float *const mB, 
				       float *const mR, const int n);


/*
  Función para sumar dos vectores en la GPU *r* veces
*/
void add_arrays_GPU(const float arrayA[], const float arrayB[], 
		    float arrayR[], const unsigned int n, const unsigned int blk_size, 
		    resnfo *const start, resnfo *const end)
{

  // Número de bytes de cada uno de nuestros vectores
  unsigned int numBytes = n * sizeof(float);

  // Reservamos memoria global del device (GPU) para nuestros 
  // arrays y los copiamos
  float *cA;
  hipMalloc((void **) &cA, numBytes);
  hipMemcpy(cA, arrayA, numBytes, hipMemcpyHostToDevice); // CPU -> GPU

  float *cB;
  hipMalloc((void **) &cB, numBytes);
  hipMemcpy(cB, arrayB, numBytes, hipMemcpyHostToDevice); // CPU -> GPU

  float *cR;
  hipMalloc((void **) &cR, numBytes);
  hipMemset(cR, 0, numBytes); // Inicializamos (a 0) array para el resultado

  // Bloque unidimensional de hilos (*blk_size* hilos)
  dim3 dimBlock(blk_size);

  // Rejilla unidimensional (*ceil(n/blk_size)* bloques)
  dim3 dimGrid((n + dimBlock.x - 1) / dimBlock.x);

  // Lanzamos ejecución del kernel en la GPU *r* veces
  timestamp(start);            // Medimos tiempo de cálculo en GPU
     add_arrays_kernel_cuda<<<dimGrid, dimBlock>>>(cA, cB, cR, n);
  
 
  timestamp(end);

  hipMemcpy(arrayR, cR, numBytes, hipMemcpyDeviceToHost); // GPU -> CPU

  hipFree (cA);
  hipFree (cB);
  hipFree (cR);
}


// Declaración de función para ver recursos del device
void devicenfo(void);


// Declaración de función para comprobar y ajustar los parámetros de
// ejecución del kernel a las restricciones de la GPU
void checkparams(unsigned int *n, unsigned int *cb);


/*
  Función principal
*/
int main(int argc, char *argv[])
{
  // Para medir tiempos
  resnfo start, end, startgpu, endgpu;
  timenfo time, timegpu;

  // Aceptamos algunos parámetros

  // Número de elementos en los vectores (predeterminado: N)
  unsigned int n = (argc > 1)?atoi (argv[1]):N;

  if (n == 0) {
    devicenfo();
    return(0);
  }

   // Número de hilos en cada bloque CUDA (predeterminado: CUDA_BLK)
  unsigned int cb = (argc > 2)?atoi (argv[2]):CUDA_BLK;

  checkparams(&n, &cb);

  // Número de bytes a reservar para nuestros vectores
  unsigned int numBytes = n * sizeof(float);

  // Reservamos e inicializamos vectores
  timestamp(&start);
  float *vectorA = (float *) malloc(numBytes);
  float *vectorB = (float *) malloc(numBytes);
  float *vectorR = (float *) malloc(numBytes);
  populating_arrays(vectorA, vectorB, vectorR, n);
  timestamp(&end);

  myElapsedtime(start, end, &time);
  printtime(time);
  printf(" -> Reservar e inicializar vectores (%u)\n\n", n);


  // Sumamos vectores en CPU
  timestamp(&start);
  add_arrays_CPU(vectorA, vectorB, vectorR, n);
  timestamp(&end);

  myElapsedtime(start, end, &time);
  printtime(time);
  printf(" -> Sumar vectores en CPU \n\n");

  // Sumamos elementos de vector resultante, para comprobar cálculo en GPU
  float result = checkini_array(vectorR, n);


  // Sumamos vectores en GPU
  timestamp(&start);
  add_arrays_GPU(vectorA, vectorB, vectorR, n, cb, &startgpu, &endgpu);
  timestamp(&end);

  // Sumamos elementos de vector resultante, para comprobar cálculo en GPU
  float result_gpu = checkini_array(vectorR, n);

  myElapsedtime(start, end, &time);
  printtime(time);
  printf(" -> Sumar vectores en GPU (%d hilos, %d hilos/bloq)\n", n, cb);
  if (result_gpu == result) // Comprobamos si resultado numérico es OK
    printf("\t\t      Resultado de la suma OK\n\n");
  else
    printf("\t\t      mec!\n\n");

  // Separamos tiempo de cálculo en GPU de tiempo de transferencia
  myElapsedtime(startgpu, endgpu, &timegpu);
  printf("\t\tDesglose:\n\t\t");	
  printtime(timegpu);
  printf("tiempo cálculo en GPU\n\t\t%15f s alloc y comm\n", time - timegpu);

  free(vectorA);
  free(vectorB);
  free(vectorR);

  return(0);
}


/*
  Definición de nuestro kernel para sumar dos vectores en CUDA
*/
__global__ void add_arrays_kernel_cuda(const float *const mA, 
				       const float *const mB, 
				       float *const mR, const int n)
{
  int global_id = blockIdx.x * blockDim.x + threadIdx.x;

   if (global_id < n)
    mR[global_id] += mA[global_id] + mB[global_id];
}


/*
  Sacar por pantalla información del *device*
*/
void devicenfo(void)
{
  struct hipDeviceProp_t capabilities;

  hipGetDeviceProperties (&capabilities, 0);

  printf("->CUDA Platform & Capabilities\n");
  printf("Name: %s\n", capabilities.name);
  printf("totalGlobalMem: %.2f MB\n", capabilities.totalGlobalMem/1024.0f/1024.0f);
  printf("sharedMemPerBlock: %.2f KB\n", capabilities.sharedMemPerBlock/1024.0f);
  printf("regsPerBlock (32 bits): %d\n", capabilities.regsPerBlock);
  printf("warpSize: %d\n", capabilities.warpSize);
  printf("memPitch: %.2f KB\n", capabilities.memPitch/1024.0f);
  printf("maxThreadsPerBlock: %d\n", capabilities.maxThreadsPerBlock);
  printf("maxThreadsDim: %d x %d x %d\n", capabilities.maxThreadsDim[0], 
	 capabilities.maxThreadsDim[1], capabilities.maxThreadsDim[2]);
  printf("maxGridSize: %d x %d\n", capabilities.maxGridSize[0], 
	 capabilities.maxGridSize[1]);
  printf("totalConstMem: %.2f KB\n", capabilities.totalConstMem/1024.0f);
  printf("major.minor: %d.%d\n", capabilities.major, capabilities.minor);
  printf("clockRate: %.2f MHz\n", capabilities.clockRate/1024.0f);
  printf("deviceOverlap: %d\n", capabilities.deviceOverlap);
  printf("multiProcessorCount: %d\n", capabilities.multiProcessorCount);
}


/*
  Función que ajusta el número de hilos, de bloques, y de bloques por hilo 
  de acuerdo a las restricciones de la GPU
*/
void checkparams(unsigned int *n, unsigned int *cb)
{
  struct hipDeviceProp_t capabilities;

  // Si menos numero total de hilos que tamaño bloque, reducimos bloque
  if (*cb > *n)
    *cb = *n;

  hipGetDeviceProperties (&capabilities, 0);

  if (*cb > capabilities.maxThreadsDim[0]) {
    *cb = capabilities.maxThreadsDim[0];
    printf("->Núm. hilos/bloq cambiado a %d (máx por bloque para dev)\n\n", 
	   *cb);
  }

  if (((*n + *cb - 1) / *cb) > capabilities.maxGridSize[0]) {
    *cb = 2 * (*n - 1) / (capabilities.maxGridSize[0] - 1);
    if (*cb > capabilities.maxThreadsDim[0]) {
      *cb = capabilities.maxThreadsDim[0];
      printf("->Núm. hilos/bloq cambiado a %d (máx por bloque para dev)\n", 
	     *cb);
      if (*n > (capabilities.maxGridSize[0] * *cb)) {
	*n = capabilities.maxGridSize[0] * *cb;
	printf("->Núm. total de hilos cambiado a %d (máx por grid para \
dev)\n\n", *n);
      } else {
	printf("\n");
      }
    } else {
      printf("->Núm. hilos/bloq cambiado a %d (%d máx. bloq/grid para \
dev)\n\n", 
	     *cb, capabilities.maxGridSize[0]);
    }
  }
}
